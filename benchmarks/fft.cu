#include <stdexcept>
#include <thrust/device_vector.h>
#include <vector>
#include <complex>
#include <hipfft/hipfft.h>

using cd = std::complex<double>;

auto cuFFT(const std::vector<cd>& data,const int N)
{
    
    
    thrust::device_vector<cd> datain;
    
    thrust::copy(data.begin(),data.begin()+N, std::back_inserter(datain) );
    
    hipfftDoubleComplex * dataptr = reinterpret_cast<hipfftDoubleComplex*>(
        thrust::raw_pointer_cast(&datain[0]));
    hipfftHandle plan;
    
    if(hipfftPlan1d(&plan,datain.size(),HIPFFT_Z2Z,1)!=HIPFFT_SUCCESS)
    {
        throw std::runtime_error("Cuda error: plan creation failed");
    }
    
    if(hipfftExecZ2Z(plan,dataptr,dataptr,HIPFFT_FORWARD)!=HIPFFT_SUCCESS)
    {
        throw std::runtime_error("Cuda error: ExecZ2Z failed");
    }
    
    if(hipDeviceSynchronize()!=hipSuccess)
    {   
        throw std::runtime_error("Cuda error: failed to syncronize");
    }
    hipfftDestroy(plan);
    
    std::vector<cd> dataout;
    thrust::copy(datain.begin(),datain.end(), std::back_inserter(dataout) );
    return dataout;
}
